#include "hip/hip_runtime.h"

#include <fstream>
#include <iostream>

class Student
{
public:
	Student() {}
	Student(std::string _name, int year, float grade, char gender)
		: year(year), grade(grade), gender(gender)
	{
		strcpy(name, _name.c_str());
		sprintf(c_year, "%d", year);
		sprintf(c_grade, "%f", grade);
	}
	
	char name[20];
	int year;
	char c_year[10];
	float grade;
	char c_grade[10];
	char gender;
};

void read_data (const char* filePath, Student students[]) {

	std::ifstream fin(filePath);

	size_t index = 0;
	while (!fin.eof())
	{
		std::string name;
		int year;
		float grade;
		char gender;
		fin >> name >> year >> grade >> gender >> std::ws;

		students[index++] = Student(name, year, grade, gender);
	}
	fin.close();
}

__global__ void process_data(Student *device_students,char *device_results, int* result_space, int *write_index);

int main() {

	Student students[1000];
	read_data("data2.csv", students);	

	int result_space = 30;
	int write_index = 0;

	// Result string
	char *host_results = new char[sizeof(char) * result_space * 1000];

	// Allocate GPU memory
	Student *device_students;
	char *device_results;
	int *device_result_space;
	int *device_write_index;

	hipMalloc((void**) &device_results		, sizeof(char) * result_space * 1000);
	hipMalloc((void**) &device_students	, sizeof(Student) * 1000);
	hipMalloc((void**) &device_result_space, sizeof(int));
	hipMalloc((void**) &device_write_index , sizeof(int));

	// Copy from CPU to GPU data that is needed
	hipMemcpy(device_students, &students[0], sizeof(Student) * 1000, hipMemcpyHostToDevice);
	hipMemcpy(device_result_space, &result_space, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_write_index , &write_index,   sizeof(int), hipMemcpyHostToDevice);

	// Run
	process_data<<<1, 499>>>(device_students, device_results, device_result_space, device_write_index);
	hipDeviceSynchronize();


	auto err = hipMemcpy(host_results, device_results, sizeof(char) * result_space * 1000, hipMemcpyDeviceToHost); // copy students to GPU
	std::cout << "Copy to host "<< err << std::endl;
	// std::cout << "Result: \n"<< host_results << std::endl;

	std::cout << "Writting results\n";
	std::ofstream fout("rez.txt");

	fout << host_results; 

	fout.close();
	std::cout << "Finished writting results\n";

	// Fee CPU and GPU memory
	free(host_results);
	hipFree(device_results);
	hipFree(device_students);
	hipFree(device_result_space);
}

__global__ void process_data(Student *device_students,char *device_results, int *result_space, int *write_index) {

	// Calculate the working index range
	const auto work_block = 1000 / blockDim.x;
	int start_index = work_block * threadIdx.x;
	int end_index;

	if (threadIdx.x == blockDim.x - 1) {
		end_index = 1000;
	}
	else {
		end_index = work_block * (threadIdx.x + 1);
	}

	// printf("Thread count: %d\nThread nr: %d\nThread start_index: %d\nThread end_index: %d\nThread work_block: %d\n\n",blockDim.x, threadIdx.x, start_index, end_index, work_block);

	for (auto i = start_index; i < end_index; i++)
	{
		auto student = device_students[i];
		long hash;
		long mul = 1;

		for (size_t d = 0; d < 10000; d++)
		{
			for (int h = 0; h != 20 ; h++)
			{
				mul = (h % 4 == 0) ? 1 : mul * 256 * i;
				hash += student.name[h] * mul;
			}
		}

		if( hash < 0){
			hash = hash * -1;
		}
		// printf("%d %d %s\n",threadIdx.x, i, student.name);
		char buffer[100];
		
		int current_index = 0;
		for (size_t f = 0; student.name[f] != '\0'; f++)
		{
			buffer[current_index++] = student.name[f];
		}
		buffer[current_index++] = '-';
		int year_index = current_index;
		for (size_t f = 0; student.c_year[f] != '\0'; f++)
		{
			buffer[current_index++] = student.c_year[f];
		}
		buffer[current_index++] = '-';
		for (size_t f = 0; f < 3; f++)
		{
			buffer[current_index++] = student.c_grade[f];
		}
		buffer[current_index++] = '-';
		
		buffer[current_index++] = '|';
		int break_counter = 7;
		for (size_t i = 0; i < hash && break_counter > 0; i+= 255)
		{
			int s = hash / (i + 1); 
			buffer[current_index++] = (char)((s % 125)+33);
			break_counter--;
		}
		buffer[current_index++] = '|';

		// printf("Thread id: %d - %d\n",threadIdx.x,  hash);

		if ((buffer[year_index] - 48) > 2) { // Filter 3, 4
			// printf("%d %s\n", threadIdx.x, buffer);
			int offset = atomicAdd(write_index, 1) * (*result_space);
			bool buffer_ended = false;
			for (size_t j = 0; j < *result_space; j++)
			{
				if(buffer[j] == '\0')
					buffer_ended = true;
				device_results[j + offset] = buffer_ended ? ' ' : buffer[j];
			}
		}




		
	}
	

	
}
